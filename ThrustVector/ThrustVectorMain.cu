#include "hip/hip_runtime.h"
#include<iostream>

#include<thrust/device_vector.h>
#include<thrust/tuple.h>
#include<thrust/transform.h>
#include<thrust/iterator/zip_iterator.h>

// 3�����x�N�g���i�̃^�v���j
typedef thrust::tuple<double, double, double> Double3;

// 2�����x�N�g���̑傫�����擾����
struct GetLength3 : public thrust::unary_function<const Double3, double>
{
	__host__ __device__
	double operator()(const Double3& v) const
	{
		double x = v.get<0>();
		double y = v.get<1>();
		double z = v.get<2>();

		return std::sqrt(x*x + y*y + z*z);
	}
} getLength3;

// 3�����x�N�g��Thrust
void Length3Thrust()
{
	// �v�f��
	const int N = 5;

	// x, y, z��������
	double x[N] = {0, 1, 2, 3, 4};
	double y[N] = {1, 2, 3, 4, 5};
	double z[N] = {2, 3, 4, 5, 6};

	// �e�x�N�g���̑傫��
	double length[N];

	// �f�o�C�X�̔z��𐶐�
	thrust::device_vector<double> xVector(x, x + N);
	thrust::device_vector<double> yVector(y, y + N);
	thrust::device_vector<double> zVector(z, z + N);
	thrust::device_vector<double> lengthVector(N);

	// �^�v��������āA���̃^�v���̃C�e���[�^�[���쐬
	auto double3Tuple = thrust::make_tuple(xVector.begin(), yVector.begin(), zVector.begin());
	auto double3Iterator = thrust::make_zip_iterator(double3Tuple);
	
	// �傫�����v�Z
	thrust::transform(double3Iterator, double3Iterator + N,
		lengthVector.begin(),
		getLength3);
	thrust::copy_n(lengthVector.begin(), N, length);

	// ���ʂ�\��
	/*
	* 2.23, 3.74, 5.39, 7.07, 7.77���炢
	*/
	std::cout << "3�����x�N�g��CPU" << std::endl;
	for(int i = 0; i < N; i++)
	{
		std::cout << i << ": " << length[i] << std::endl;
	}
}


// 3�����x�N�g��CPU
void Length3()
{
	// �v�f��
	const int N = 5;

	// x, y, z��������
	double x[N] = {0, 1, 2, 3, 4};
	double y[N] = {1, 2, 3, 4, 5};
	double z[N] = {2, 3, 4, 5, 6};

	// �e�x�N�g���̑傫��
	double length[N];

	// �傫�����v�Z
	for(int i = 0; i < N; i++)
	{
		// ��(x^2 + y^2 + z^2)
		length[i] = std::sqrt(x[i]*x[i] + y[i]*y[i] + z[i]*z[i]);
	}

	// ���ʂ�\��
	/*
	* 2.23, 3.74, 5.39, 7.07, 7.77���炢
	*/
	std::cout << "3�����x�N�g��CPU" << std::endl;
	for(int i = 0; i < N; i++)
	{
		std::cout << i << ": " << length[i] << std::endl;
	}
}

// 2�����x�N�g���̑傫�����擾����
struct GetLength2 : public thrust::binary_function<const double, const double, double>
{
	__host__ __device__
	double operator()(const double& x, const double& y) const
	{
		return std::sqrt(x*x + y*y);
	}
} getLength2;

// 2�����x�N�g��Thrust
void Length2Thrust()
{
	// �v�f��
	const int N = 5;

	// x, y��������
	double x[N] = {0, 1, 2, 3, 4};
	double y[N] = {1, 2, 3, 4, 5};

	// �e�x�N�g���̑傫��
	double length[N];

	// �f�o�C�X�̔z��𐶐�
	thrust::device_vector<double> xVector(x, x + N);
	thrust::device_vector<double> yVector(y, y + N);
	thrust::device_vector<double> lengthVector(N);

	// �傫�����v�Z
	thrust::transform(xVector.begin(), xVector.begin() + N, yVector.begin(), lengthVector.begin(), getLength2);
	thrust::copy_n(lengthVector.begin(), N, length);

	// ���ʂ�\��
	/*
	* 1.00, 2.24, 3.61, 5.00, 6.40���炢
	*/
	std::cout << "2�����x�N�g��Thrust" << std::endl;
	for(int i = 0; i < N; i++)
	{
		std::cout << i << ": " << length[i] << std::endl;
	}
}

// 2�����x�N�g��CPU
void Length2()
{
	// �v�f��
	const int N = 5;

	// x, y��������
	double x[N] = {0, 1, 2, 3, 4};
	double y[N] = {1, 2, 3, 4, 5};

	// �e�x�N�g���̑傫��
	double length[N];

	// �傫�����v�Z
	for(int i = 0; i < N; i++)
	{
		// ��(x^2 + y^2 + z^2)
		length[i] = std::sqrt(x[i]*x[i] + y[i]*y[i]);
	}

	// ���ʂ�\��
	/*
	* 1.00, 2.24, 3.61, 5.00, 6.40���炢
	*/
	std::cout << "2�����x�N�g��CPU" << std::endl;
	for(int i = 0; i < N; i++)
	{
		std::cout << i << ": " << length[i] << std::endl;
	}
}

// �G���g���|�C���g
int main()
{
	// Thrust
	Length2Thrust();
	Length3Thrust();

	// CPU
	Length2();
	Length3();

	return 0;
}